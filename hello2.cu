#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello(int totalThreads)
{
  int myID = (blockIdx.x * blockDim.x) + threadIdx.x;
  int globalID = myID + blockIdx.x * blockDim.x * gridDim.x;

  if (myID < totalThreads)
  {
    if (myID == totalThreads - 1)
    {
      printf("Hello world from the thread with the maximum global ID: %i (global ID: %i)\n", myID, globalID);
    }
  }
}

int main()
{
  int totalThreads = 100000;  // Número total de hilos
  int threadsPerBlock = 256; // Número de hilos por bloque (puedes ajustarlo según tu preferencia)

  int blocks = (totalThreads + threadsPerBlock - 1) / threadsPerBlock;  // Calcula el número de bloques necesario

  dim3 g(blocks, 1, 1);  // Dimensión de la grilla
  dim3 b(threadsPerBlock, 1, 1);   // Dimensión de los bloques

  hello <<<g, b>>>(totalThreads);
  hipDeviceSynchronize();

  return 0;
}

